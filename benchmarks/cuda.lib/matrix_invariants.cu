
#include <hip/hip_runtime.h>
// -*- c++ -*-
//
// sebastiaan van paasen
// (c) 1998-2023 all rights reserved

// function to compute the invariants of a 3x3 tensor
__global__ void
computeInvariants(const double * A, double * I1, double * I2, double * I3, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        double A00 = A[index];
        double A01 = A[index + size];
        double A02 = A[index + 2 * size];
        double A10 = A[index + 3 * size];
        double A11 = A[index + 4 * size];
        double A12 = A[index + 5 * size];
        double A20 = A[index + 6 * size];
        double A21 = A[index + 7 * size];
        double A22 = A[index + 8 * size];

        // compute the first invariant
        I1[index] = A00 + A11 + A22;

        // compute the second invariant
        I2[index] = A00 * A11 + A11 * A22 + A00 * A22 - A01 * A10 - A02 * A20 - A12 * A21;

        // compute the third invariant
        I3[index] = A00 * (A11 * A22 - A12 * A21) - A01 * (A10 * A22 - A12 * A20)
                  + A02 * (A10 * A21 - A11 * A20);
    }

    return;
}

void
computeInvariantsManaged(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * I1,
    double * I2, double * I3)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(tensorArray, I1, I2, I3, nTensors);

    // all done
    return;
}

void
computeInvariantsPinned(
    int nTensors, int nThreadPerBlock, int nBlocks, double * gpuTensors, double * gpuI1,
    double * gpuI2, double * gpuI3)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(gpuTensors, gpuI1, gpuI2, gpuI3, nTensors);

    // all done
    return;
}

void
computeInvariantsMapped(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * I1,
    double * I2, double * I3)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(tensorArray, I1, I2, I3, nTensors);

    // all done
    return;
}

// end of file
