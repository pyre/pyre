
#include <hip/hip_runtime.h>
// -*- c++ -*-
//
// sebastiaan van paasen
// (c) 1998-2023 all rights reserved

// function to compute the invariants of a 3x3 tensor
__global__ void
computeInverse(const double * A, double * Ainv, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        double A00 = A[index];
        double A01 = A[index + size];
        double A02 = A[index + 2 * size];
        double A10 = A[index + 3 * size];
        double A11 = A[index + 4 * size];
        double A12 = A[index + 5 * size];
        double A20 = A[index + 6 * size];
        double A21 = A[index + 7 * size];
        double A22 = A[index + 8 * size];

        // compute the determinant of A
        double detA = A00 * (A11 * A22 - A12 * A21) - A01 * (A10 * A22 - A12 * A20)
                    + A02 * (A10 * A21 - A11 * A20);

        // set to its inverse
        double detInv = 1.0 / detA;

        // and return the inverse tensor
        Ainv[index] = detInv * (A11 * A22 - A12 * A21);
        Ainv[index + size] = detInv * (-A01 * A22 + A02 * A21);
        Ainv[index + 2 * size] = detInv * (A01 * A12 - A02 * A11);
        Ainv[index + 3 * size] = detInv * (-A10 * A22 + A12 * A20);
        Ainv[index + 4 * size] = detInv * (A00 * A22 - A02 * A20);
        Ainv[index + 5 * size] = detInv * (-A00 * A12 + A02 * A10);
        Ainv[index + 6 * size] = detInv * (A10 * A21 - A11 * A20);
        Ainv[index + 7 * size] = detInv * (-A00 * A21 + A01 * A20);
        Ainv[index + 8 * size] = detInv * (A00 * A11 - A01 * A10);
    }

    return;
}

void
computeInvariantsManaged(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray,
    double * inverseArray)
{
    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(tensorArray, inverseArray, nTensors);

    // all done
    return;
}


void
computeInvariantsPinned(
    int nTensors, int nThreadPerBlock, int nBlocks, double * gpuTensors, double * gpuInverses)
{
    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(gpuTensors, gpuInverses, nTensors);

    // all done
    return;
}

void
computeInvariantsMapped(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray,
    double * inverseArray)
{
    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(tensorArray, inverseArray, nTensors);

    // all done
    return;
}

// end of file
